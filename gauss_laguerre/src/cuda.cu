#include "hip/hip_runtime.h"
#include "../include/common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>

__global__ void nc_compute_kernel(
    int n, double x_min, double x_max, const double* x, double* w
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    extern __shared__ double d[];

    for (int j = 0; j < n; j++) d[j] = 0.0;
    d[i] = 1.0;

    for (int j = 2; j <= n; j++) {
        for (int k = j; k <= n; k++)
            d[n + j - k - 1] = (d[n + j - k - 2] - d[n + j - k - 1]) /
                               (x[n - k] - x[n + j - k - 1]);
    }
    for (int j = 1; j <= n - 1; j++) {
        for (int k = 1; k <= n - j; k++)
            d[n - k - 1] -= x[n - k - j] * d[n - k];
    }

    double yvala = d[n - 1] / n, yvalb = d[n - 1] / n;
    for (int j = n - 2; j >= 0; j--) {
        yvala = yvala * x_min + d[j] / (j + 1);
        yvalb = yvalb * x_max + d[j] / (j + 1);
    }
    w[i] = yvalb * x_max - yvala * x_min;
}

double *nc_compute_new(int n, double x_min, double x_max, double x[]) {
    double *h_w = (double*)malloc(n * sizeof(double));
    double *d_x, *d_w;

    hipMalloc(&d_x, n * sizeof(double));
    hipMalloc(&d_w, n * sizeof(double));
    hipMemcpy(d_x, x, n * sizeof(double), hipMemcpyHostToDevice);

    int threads = 128;
    int blocks = (n + threads - 1) / threads;
    nc_compute_kernel<<<blocks, threads, n * sizeof(double)>>>(n, x_min, x_max, d_x, d_w);
    hipDeviceSynchronize();

    hipMemcpy(h_w, d_w, n * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_x);
    hipFree(d_w);
    return h_w;
}

__global__ void extract_and_linearscale(hipfftDoubleComplex *Y, double *w, int n, double a, double b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    double scale = 2.0 / n;
    if (i == 0 || i == n - 1) scale *= 0.5;
    w[i] = scale * Y[i].x * ((b - a) * 0.5);
}

double *nc_compute_new_fft(int n, double a, double b, double x[]) {
    int nfft = 2 * n;
    double *d_y;
    hipfftDoubleComplex *d_Y;
    double *h_w;

    hipHostMalloc(&h_w, n * sizeof(double));
    hipMalloc(&d_y, nfft * sizeof(double));
    hipMalloc(&d_Y, (n + 1) * sizeof(hipfftDoubleComplex));

    double *h_y = (double *)malloc((n + 1) * sizeof(double));
    h_y[0] = 1.0;
    h_y[n] = (n & 1) ? -1.0 : 1.0;
    for (int k = 1; k < n; ++k)
        h_y[k] = x[k] + x[n - k];

    hipMemcpy(d_y, h_y, (n + 1) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_y + n + 1, h_y + 1, (n - 1) * sizeof(double), hipMemcpyHostToDevice);
    free(h_y);

    hipfftHandle plan;
    hipfftPlan1d(&plan, nfft, HIPFFT_D2Z, 1);
    hipfftExecD2Z(plan, d_y, d_Y);

    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    extract_and_linearscale<<<blocks, threads>>>(d_Y, h_w, n, a, b);
    hipDeviceSynchronize();

    hipfftDestroy(plan);
    hipFree(d_y);
    hipFree(d_Y);

    return h_w;
}

int main(int argc, char *argv[]) {
    double a, b;
    int n;
    char out_prefix[256];
    if (argc >= 2) n = atoi(argv[1]); else { printf("Enter N: "); scanf("%d", &n); }
    if (argc >= 3) a = atof(argv[2]); else { printf("Enter A: "); scanf("%lf", &a); }
    if (argc >= 4) b = atof(argv[3]); else { printf("Enter B: "); scanf("%lf", &b); }
    if (argc >= 5) strncpy(out_prefix, argv[4], 255); else { printf("Enter root filename: "); scanf("%s", out_prefix); }
    out_prefix[255] = '\0';

    char base[256];
    get_output_base(out_prefix, base, sizeof(base));

    double *x_ref = (double *)malloc(n * sizeof(double));
    double *w_ref = (double *)malloc(n * sizeof(double));
    if (!load_sequential_result(base, n, "x", x_ref) || !load_sequential_result(base, n, "w", w_ref)) {
        fprintf(stderr, "Failed to load precomputed x or w files.\n");
        exit(EXIT_FAILURE);
    }

    double seq_time = 0.0;
    if (!load_sequential_timing(base, &seq_time)) {
        fprintf(stderr, "No times found in sequential timing file for %s\n", base);
        exit(EXIT_FAILURE);
    }

    double *r = (double *)malloc(2 * sizeof(double));
    r[0] = a; r[1] = b;

    double *x_calc = ccn_compute_points_new(n);
    double *w_warmup = nc_compute_new(n, a, b, x_calc);
    hipHostFree(w_warmup);
    hipEvent_t start, stop; hipEventCreate(&start); hipEventCreate(&stop);
    hipEventRecord(start);
    double *w_calc = nc_compute_new(n, a, b, x_calc);
    hipEventRecord(stop); hipEventSynchronize(stop);
    float par_time;
    hipEventElapsedTime(&par_time, start, stop);
    par_time /= 1000.0;

    for (int i = 0; i < n; i++)
        x_calc[i] = ((a + b) + (b - a) * x_calc[i]) * 0.5;

    int ok = compare_results(x_ref, x_calc, n, 1e-6) && compare_results(w_ref, w_calc, n, 1e-6);

    printf("\n");
    printf("%s  Test %s%s\n", BOLD, ok ? GREEN "PASSED" : RED "FAILED", CLEAR);
    printf("%s  Sequential time: %s%.6fs %s\n", BOLD, BLUE, seq_time, CLEAR);
    printf("%s  Parallel time:   %s%.6fs %s\n", BOLD, BLUE, par_time, CLEAR);
    printf("%s  Speedup:         %s%.3fx %s\n", BOLD, BLUE, seq_time / par_time, CLEAR);
    printf("\n");
    rule_write(n, out_prefix, x_calc, w_calc, r);
    append_timing(out_prefix, par_time);

    free(r);
    free(x_ref); free(w_ref);
    free(x_calc);
    if (w_calc)
        hipHostFree(w_calc);

    hipEventDestroy(start); hipEventDestroy(stop);
    return 0;
}
